#include "hip/hip_runtime.h"

//#include <openssl/sha.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <sys/time.h>
#include "sha256.cu"
 
#define SHA256_DIGEST_SIZE 32
#define repeats  1000
#define NUM_BLOCKS 1024


typedef struct block_header {
        unsigned int    version;
        
        unsigned char   prev_block[32];
        unsigned char   merkle_root[32];
        unsigned int    timestamp;
        unsigned int    bits;
        unsigned int    nonce;
} block_header;
 
double When()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) tp.tv_sec + (double) tp.tv_usec * 1e-6);
}


void hex2bin(unsigned char* dest, const char* src)
{
        int c, pos;
        char buf[3];
 
        pos=0;
        c=0;
        buf[2] = 0;
        while(c < strlen(src))
        {

                buf[0] = src[c++];
                buf[1] = src[c++];

                dest[pos++] = (unsigned char)strtol(buf, NULL, 16);
        }
       
}
 

__device__ void print_hash(unsigned char hash[])
{
   int idx;
   for (idx=0; idx < 32; idx++)
      printf("%02x",hash[idx]);
   printf("\n");
}
 

__device__ __host__ void byte_swap(unsigned char* data) {
        int c;
        unsigned char tmp[SHA256_DIGEST_SIZE];
       
        c=0;
        while(c<SHA256_DIGEST_SIZE)
        {
                tmp[c] = data[SHA256_DIGEST_SIZE-(c+1)];
                c++;
        }
       
        c=0;
        while(c<SHA256_DIGEST_SIZE)
        {
                data[c] = tmp[c];
                c++;
        }
}

__global__ void doCalc(unsigned char *dev_prev_block, unsigned char *dev_merkle_root, int seed) {
    int i;   
    block_header header;
    header.version =        2;
    header.timestamp =      1392872245;
    header.bits =           419520339;

    for(i=0;i<32;i++) {
        header.prev_block[i] = dev_prev_block[i];
        header.merkle_root[i] = dev_merkle_root[i];

    }



    unsigned char hash1[32];
    unsigned char hash2[32];
   

    SHA256_CTX sha256_pass1, sha256_pass2;

    header.nonce = (seed *  blockDim.x * NUM_BLOCKS) + blockIdx.x * blockDim.x + threadIdx.x*repeats;
  

    for(i=0;i<repeats;i++) {
        sha256_init(&sha256_pass1);

        sha256_update(&sha256_pass1, (unsigned char*)&header, sizeof(block_header));

        sha256_final(&sha256_pass1,hash1);
           
         
        sha256_init(&sha256_pass2);
        sha256_update(&sha256_pass2, hash1, SHA256_DIGEST_SIZE);
        sha256_final(&sha256_pass2, hash2);
         if ( header.nonce == 0 || header.nonce == 3 || header.nonce == 856192328 ) {
             //hexdump((unsigned char*)&header, sizeof(block_header));
             //printf("%u:\n", header.nonce);
             byte_swap(hash2);
             //printf("Target Second Pass Checksum: \n");
             //print_hash(hash2);
         }
        header.nonce++;
    }
}

int main() {
    int i = 0;
    int blocksize = 16;
    int threads = 128;

    long long hashes = 0;

    int counter = 0;

    unsigned char *dev_merkle_root, *dev_prev_block;

    unsigned char prev_block[32], merkle_root[32];

    hex2bin(prev_block,              "000000000000000117c80378b8da0e33559b5997f2ad55e2f7d18ec1975b9717");
    hex2bin(merkle_root,             "871714dcbae6c8193a2bb9b2a69fe1c0440399f38d94b3a0f1b447275a29978a");


    byte_swap(prev_block);
    byte_swap(merkle_root);

   

    hipMalloc((void**)&dev_prev_block, 32*sizeof(unsigned char));
    hipMemcpy(dev_prev_block, &(prev_block), 32 * sizeof(unsigned char), hipMemcpyHostToDevice);

    hipMalloc((void**)&dev_merkle_root, 32*sizeof(unsigned char));
    hipMemcpy(dev_merkle_root, &(merkle_root), 32 * sizeof(unsigned char), hipMemcpyHostToDevice);

    double start = When();
    double timer = When() - start;
    while ( timer < 60.0){

        doCalc<<< blocksize, threads >>>(dev_prev_block, dev_merkle_root, counter);
        hashes += blocksize*threads*repeats;
        counter++;
        timer = When() - start;
   
        hipDeviceSynchronize();
    }
 
    printf("number of hashs per second = %lld\n",(long long) (hashes / (When() - start)) );

 
    return 0;
}
